#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace cv;
using namespace std;

__device__ void rgb_to_hsv(unsigned char r, unsigned char g, unsigned char b, float &h, float &s, float &v) {
    float fr = r / 255.0f;
    float fg = g / 255.0f;
    float fb = b / 255.0f;

    float cmax = fmaxf(fr, fmaxf(fg, fb));
    float cmin = fminf(fr, fminf(fg, fb));
    float delta = cmax - cmin;

    h = 0.0f;

    if (delta != 0.0f) {
        if (cmax == fr) {
            h = 60.0f * fmodf(((fg - fb) / delta), 6.0f);
        } else if (cmax == fg) {
            h = 60.0f * (((fb - fr) / delta) + 2.0f);
        } else {
            h = 60.0f * (((fr - fg) / delta) + 4.0f);
        }
    }

    if (h < 0.0f) h += 360.0f;

    s = (cmax == 0.0f) ? 0.0f : delta / cmax;
    v = cmax;
}

__global__ void hsv_chroma_kernel(uchar3* input, uchar3* bg, uchar3* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;

    uchar3 pixel = input[idx];
    float h, s, v;
    rgb_to_hsv(pixel.x, pixel.y, pixel.z, h, s, v);

    bool is_green = (h >= 60.0f && h <= 180.0f && s > 0.4f && v > 0.3f);
    output[idx] = is_green ? bg[idx] : pixel;
}

int main() {
    VideoCapture cap(0);
    if (!cap.isOpened()) {
        cerr << "Failed to open webcam!" << endl;
        return -1;
    }

    Mat frame, background, resized_bg;
    cap >> frame;
    int width = frame.cols;
    int height = frame.rows;

    // Load replacement background image
    background = imread("background.jpg");
    if (background.empty()) {
        cerr << "Failed to load background.jpg!" << endl;
        return -1;
    }
    resize(background, resized_bg, Size(width, height));

    // Allocate host/device memory
    uchar3 *d_input, *d_bg, *d_output;
    size_t total = width * height * sizeof(uchar3);

    hipMalloc(&d_input, total);
    hipMalloc(&d_bg, total);
    hipMalloc(&d_output, total);

    Mat output_frame(height, width, CV_8UC3);

    while (true) {
        cap >> frame;
        if (frame.empty()) break;

        resize(background, resized_bg, frame.size());

        hipMemcpy(d_input, frame.ptr<uchar3>(), total, hipMemcpyHostToDevice);
        hipMemcpy(d_bg, resized_bg.ptr<uchar3>(), total, hipMemcpyHostToDevice);

        dim3 block(16, 16);
        dim3 grid((width + 15) / 16, (height + 15) / 16);
        hsv_chroma_kernel<<<grid, block>>>(d_input, d_bg, d_output, width, height);

        hipMemcpy(output_frame.ptr<uchar3>(), d_output, total, hipMemcpyDeviceToHost);

        imshow("HSV Chroma Key", output_frame);
        if (waitKey(1) == 27) break;  // ESC key to exit
    }

    hipFree(d_input);
    hipFree(d_bg);
    hipFree(d_output);
    cap.release();
    destroyAllWindows();
    return 0;
}
